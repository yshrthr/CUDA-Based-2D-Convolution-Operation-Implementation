/**
*   CS6023: GPU Programming
*   Assignment 2
*
*   Please don't change any existing code in this file.
*
*   Please add necessary memory APIs for your implementation. Use cudaFree()
*   to free up memory as soon as you're done with an allocation.
*   This will ensure that you don't run out of memory while running
*   large test cases. Use the minimum required memory for your
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

using std::cin;
using std::cout;

typedef long long ll;
__global__
void dkernel(long int* d_mat, long int *d_filter, long int *d_ans, int* matrix_row, int* matrix_col ,int* filter_size){
  int row = blockIdx.x;
  int col = threadIdx.x;
  int k = *filter_size;
  int m = *matrix_row;
  int n = *matrix_col;

  //printf("row: %d, col: %d, %ld\n ", row, col, d_mat[row * n + col]);
  long int sum = 0;
  for(int k1 = 0, i = row - (k/2); k1 < k && i <= row + (k/2); k1++, i++){
    for(int k2 = 0, j = col - (k/2); k2 < k && j <= col + (k/2); k2++, j++){
      if(i >= 0 && i < m){
        if(j >= 0 && j < n){
          //printf("%ld, %ld\n" , d_filter[k1*k + k2], d_mat[i * n + j]);
          sum += d_filter[k1*k + k2] * d_mat[i * n + j];
        }
      }
    }
  }

  d_ans[row * n + col]  = sum;

}

int main(int argc, char** argv) {

    int m,n,k;
    cin>>m>>n>>k;


    long int* h_mat = new long int[m * n];
    long int* h_filter = new long int[k * k];

    long int* h_ans = new long int[m * n];

    long int* d_mat;
    long int *d_filter;
    long int* d_ans;
    int* filter_size;
    int* matrix_row;
    int* matrix_col;

    for (long int i = 0; i < m * n; i++) {
        cin>>h_mat[i];
    }

    for (long int i = 0; i < k * k; i++) {
        cin>>h_filter[i];
    }

    /**
     *
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     *
    **/

    /****************************************************Start Here***********************************************************/
    hipMalloc(&d_mat, (m * n) * sizeof(long int));
    hipMalloc(&d_filter, (k * k) * sizeof(long int));
    hipMalloc(&d_ans, (m * n) * sizeof(long int));
    hipMalloc(&filter_size, sizeof(int));
    hipMalloc(&matrix_row, sizeof(int));
    hipMalloc(&matrix_col, sizeof(int));


    hipMemcpy(d_mat, h_mat, m * n * sizeof(long int),hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, k * k * sizeof(long int),hipMemcpyHostToDevice);
    hipMemcpy(filter_size, &k, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(matrix_row, &m, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(matrix_col, &n, sizeof(int), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();//keep it just before the kernel launch
    //printf("launching");
    dkernel<<<m,n>>>(d_mat, d_filter, d_ans, matrix_row, matrix_col, filter_size);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();//keep it just after the kernel launch

    hipMemcpy(h_ans, d_ans, m * n * sizeof(long int), hipMemcpyDeviceToHost);

    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/
    std::chrono::duration<double> elapsed1 = end - start;
    /**
     *
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     *
    */



    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < m; i++) {
            for (long int j = 0; j < n; j++) {
                file << h_ans[i * n + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}
